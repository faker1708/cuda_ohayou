#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <unistd.h> // sleep
int main()
     {
        NvAPI_Status ret = NVAPI_OK;
        ret = NvAPI_Initialize();
 
        if(ret != NVAPI_OK)
        {
            printf("NvAPI_Initialize() failed = 0x%x", ret);
            return 1; // Initialization failed
        }
 
        NvPhysicalGpuHandle hPhysicalGpu[NVAPI_MAX_PHYSICAL_GPUS];
 
        for (NvU32 PhysicalGpuIndex = 0; PhysicalGpuIndex < NVAPI_MAX_PHYSICAL_GPUS; PhysicalGpuIndex++)
        {
            hPhysicalGpu[PhysicalGpuIndex]=0;
        }
 
         ret = NvAPI_EnumPhysicalGPUs(hPhysicalGpu, &physicalGpuCount);
        if(ret != NVAPI_OK)
        {
            printf("Cannot enumerate GPUs in the system...\n");
            getchar();
            exit(1);
        }
 
        NV_GPU_THERMAL_SETTINGS currentTemp;//获取温度的数据结构
        currentTemp.version = NV_GPU_THERMAL_SETTINGS_VER;//一定要设置，不然调用获取温度函数时候会出错
        for(NvU32 GpuIndex = 0; GpuIndex < physicalGpuCount; GpuIndex++)
        {
            ret = NvAPI_GPU_GetThermalSettings(hPhysicalGpu[PhysicalGpuIndex], NVAPI_THERMAL_TARGET_ALL,¤tTemp);//获取温度
            if(ret == NVAPI_OK)
            {
                printf("current temperature=%d\n",currentTemp.sensor[0].currentTemp);
            }
 
        }
 
        return 0;
     }