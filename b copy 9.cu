#include <iostream>
#include <vector>
#include <fstream>
#include <sstream>
#include <cstdlib>
#include <ctime>
#include <string.h>
#include <hip/hip_runtime.h>
// #include <windows.h>
#include "nvml.h"

int main(int argc, char* argv[])
{
    nvmlReturn_t result0;
    unsigned int device_count;
    result0 = nvmlInit();

    result0 = nvmlDeviceGetCount(&device_count);
    if (NVML_SUCCESS != result0)
    {
        std::cout << "Failed to query device count: " << nvmlErrorString(result0);
        return -1;
    }
    std::cout << "Found" << device_count <<" device" << std::endl;
    std::stringstream availGPUTxt;

    for (int i=0; i<device_count; i++)
    {
        nvmlDevice_t device;
        nvmlPciInfo_t pci;
        result0 = nvmlDeviceGetHandleByIndex(i, &device);
        if (NVML_SUCCESS != result0)
        {
            std::cout << "Failed to get device count: " << nvmlErrorString(result0);
            return -1;
        }

        char name[NVML_DEVICE_NAME_BUFFER_SIZE];
        result0 = nvmlDeviceGetName(device, name, NVML_DEVICE_NAME_BUFFER_SIZE);
        if (NVML_SUCCESS != result0)
        {
            std::cout << "Failed to get device name: " << nvmlErrorString(result0);
            return -1;
        }

        nvmlUtilization_t utilization;
        result0 = nvmlDeviceGetUtilizationRates(device, &utilization);
        if (NVML_SUCCESS != result0)
        {
            std::cout << "Failed to get utilization rates: " << nvmlErrorString(result0);
            return -1;
        }

        nvmlMemory_t memory;
        result0 = nvmlDeviceGetMemoryInfo(device, &memory);
        if (NVML_SUCCESS != result0)
        {
            std::cout << "Failed to get memory info: " << nvmlErrorString(result0);
            return -1;
        }

        availGPUTxt << "-----------------------------------------------------------" << std::endl;
        availGPUTxt << "CUDA NVML" << std::endl;
        availGPUTxt << "第" << i << "块显卡" << std::endl;
        availGPUTxt << "GPU name:" << name << std::endl;
        availGPUTxt << "----- 使用率: -----" << std::endl;
        availGPUTxt << "GPU使用率:" << utilization.gpu << std::endl;
        availGPUTxt << "显存使用率:" << utilization.memory << std::endl;
         // Byte->KB->MB->GB
        availGPUTxt << "全部可用显存:" << (float)(memory.total)/1024.0f/1024.0f/1024.0f << "GB" << std::endl;
        availGPUTxt << "剩余可用显存:" << (float)(memory.free)/1024.0f/1024.0f/1024.0f << "GB" << std::endl;
        availGPUTxt << "-----------------------------------------------------------" << std::endl;
    }
    
    std::string fileFitBetaName = "E:\\test\\availGPUTest.txt";
    std::ofstream fileFitBetaInfo(fileFitBetaName, std::ios::out|std::ios::binary|std::ios::ate);
    fileFitBetaInfo.write((char*)availGPUTxt.str().c_str(), (int)availGPUTxt.str().length());
    fileFitBetaInfo.close();

    return 0;
}
