
#include <hip/hip_runtime.h>
#include<stdio.h>


#include<nvml.h>


int main(){

    printf("ohayou nvml\n");
    nvmlInit_v2();
    // nvmlDeviceGetCount ();

    unsigned int device_count;

    unsigned int result;

    result = nvmlDeviceGetCount_v2(&device_count);
    if (NVML_SUCCESS != result)
    {
        printf("cannot use nvml\n");
        // printf("Failed to query device count: %s\n", nvmlErrorString(result));
        // goto Error;
    }


    printf("nvml.h\n");
}