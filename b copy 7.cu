#include <hip/hip_runtime.h>
#include<nvml.h>

#include<stdio.h>

int main(){
	hipError_t cudaStatus;
	int num;
	hipDeviceProp_t prop;
	cudaStatus = hipGetDeviceCount(&num);
	printf("deviceCount := %d\n",num);
	for(int i=0;i<num;i++){
		hipGetDeviceProperties(&prop,i);
		printf("name:%s\n",prop.name);
		printf("totalGlobalMem:%d\n",prop.totalGlobalMem);
		printf("totalGlobalMem:%d\n",prop.totalGlobalMem/1024);
		printf("totalGlobalMem:%d\n",prop.totalGlobalMem/1024/1024);
		printf("totalGlobalMem:%d\n",prop.totalGlobalMem/1024/1024/1024);
		printf("multiProcessorCount:%d\n",prop.multiProcessorCount);
		printf("maxThreadsPerBlock:%d\n",prop.maxThreadsPerBlock);
		printf("major:%d,minor:%d\n",prop.major,prop.minor);
        // printf("%d\n",prop.DeviceGetTemperature);
	}

	return 0;
}