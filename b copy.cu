#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void mykernel(void) {
    int col_index = threadIdx.x + blockIdx.x * blockDim.x;
    int row_index = threadIdx.y + blockIdx.y * blockDim.y;
    // system("pause");
    printf("hello from (%d,%d) \n",row_index,col_index);
}

int main(void) {
    dim3 grid(2,3);
    dim3 block(3,5);
    mykernel<<<grid, block>>>();
    // synchronize the device
    hipDeviceSynchronize();
}   