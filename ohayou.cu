#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

/**
 * @brief print device properties
 * 
 * @param prop 
 */
void showDeviceProp(hipDeviceProp_t &prop) {
    printf("Device name: %s\n", prop.name);
    printf("  Compute capability: %d.%d\n", prop.major, prop.minor);
    printf("  Clock rate: %d\n", prop.clockRate);
    printf("  Memory clock rate: %d\n", prop.memoryClockRate);
    printf("  Memory bus width: %d\n", prop.memoryBusWidth);
    printf("  Peak memory bandwidth: %d\n", prop.memoryBusWidth);
    printf("  Total global memory: %lu\n", prop.totalGlobalMem);
    printf("  Total shared memory per block: %lu\n", prop.sharedMemPerBlock);
    printf("  Total registers per block: %d\n", prop.regsPerBlock);
    printf("  Warp size: %d\n", prop.warpSize);
    printf("  Maximum memory pitch: %lu\n", prop.memPitch);
    printf("  Maximum threads per block: %d\n", prop.maxThreadsPerBlock);
    printf("  Maximum dimension of block: %d x %d x %d\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
    printf("  Maximum dimension of grid: %d x %d x %d\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
    printf("  Maximum memory alloc size: %lu\n", prop.totalConstMem);
    printf("  Texture alignment: %lu\n", prop.textureAlignment);
    printf("  Concurrent copy and execution: %s\n", prop.deviceOverlap ? "Yes" : "No");
    printf("  Number of multiprocessors: %d\n", prop.multiProcessorCount);
    printf("  Kernel execution timeout: %s\n", prop.kernelExecTimeoutEnabled ? "Yes" : "No");
    printf("  Integrated GPU sharing Host Memory: %s\n", prop.integrated ? "Yes" : "No");
}

int main() {
    int num_devices;
    hipDeviceProp_t properties;
    hipGetDeviceCount(&num_devices);
    printf("%d CUDA devices found\n", num_devices);
    for (int i = 0; i < num_devices; i++) {
        hipGetDeviceProperties(&properties, i);
        printf("Device %d: \"%s\"\n", i, properties.name);
        showDeviceProp(properties);
    }

    return 0;
}
